#include "hip/hip_runtime.h"
// v0.2 modified by WZ

//#include <wb.h>
//#include "/home/prof/wagner/ci853/labs/wb4.h" // use our lib instead (under construction)
//#include "/home/wagner/ci853/labs-achel/wb.h" // use our lib instead (under construction)
#include "/home/rrlmachado/labs/wb4.h" // use our lib instead (under construction)

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5

__global__ void blurImage(unsigned char *inputImage, unsigned char *outputImage, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x < width && y < height) {
    int i, j, k;

    for(k = 0; k < 3; ++k) {
      int sum = 0;
      int counter = 0;
      int anchor = BLUR_SIZE;

      for(i = x - anchor; i <= x + anchor; ++i) {
        for(j = y - anchor; j <= y + anchor; ++j) {
          if(i > -1 && i < width && j > -1 && j < height) {
            sum += inputImage[(j * width + i) * 3 + k];
            counter++;
          }
        }
      }

      outputImage[(y * width + x) * 3 + k] = (unsigned char)(sum / counter);
    }
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);
  printf( "imagem de entrada: %s\n", inputImageFile );

//  inputImage = wbImportImage(inputImageFile);
  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

// NOW: input and output images are RGB (3 channel)
  outputImage = wbImage_new(imageWidth, imageHeight, 3);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3);
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3);
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
 hipMemcpy(deviceInputImageData, hostInputImageData,
            imageWidth * imageHeight * sizeof(unsigned char) * 3,
            hipMemcpyHostToDevice);

  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  dim3 grid((imageWidth - 1) / 256 + 1, imageHeight, 1);
  dim3 block(256, 1, 1);
  
  blurImage<<<grid, block>>>(deviceInputImageData, deviceOutputImageData, imageWidth, imageHeight);
  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3,
             hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);
  // DEBUG: if you want to see your image, 
  //   will generate file bellow in current directory
  wbExport( "blurred.ppm", outputImage );

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
